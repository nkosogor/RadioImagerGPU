#include "hip/hip_runtime.h"
#include "imaging.hpp"
#include <hipfft/hipfft.h>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include <hip/hip_runtime.h>

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << " Error: " << hipGetErrorString(err) << "\n";
        exit(EXIT_FAILURE);
    }
}

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

// This atomicAdd is required if your GPU compute capability is less than 6.0
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

__global__ void mapVisibilities(hipfftDoubleComplex* grid, const hipfftDoubleComplex* visibilities, const double* u, const double* v, double uv_max, double grid_res, int image_size, int num_visibilities) {
    extern __shared__ double shared_mem[];
    double* shared_u = shared_mem;
    double* shared_v = shared_u + blockDim.x;
    hipfftDoubleComplex* shared_vis = (hipfftDoubleComplex*)(shared_v + blockDim.x);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_visibilities) {
        shared_u[threadIdx.x] = u[idx];
        shared_v[threadIdx.x] = v[idx];
        shared_vis[threadIdx.x] = visibilities[idx];
    }
    __syncthreads();

    if (idx < num_visibilities) {
        int i_index = static_cast<int>((shared_u[threadIdx.x] + uv_max) / grid_res);
        int j_index = static_cast<int>((shared_v[threadIdx.x] + uv_max) / grid_res);
        i_index = (i_index + image_size) % image_size;
        j_index = (j_index + image_size) % image_size;

        if (i_index < image_size && j_index < image_size) {
            atomicAdd(&grid[i_index * image_size + j_index].x, shared_vis[threadIdx.x].x);
            atomicAdd(&grid[i_index * image_size + j_index].y, shared_vis[threadIdx.x].y);
        }
    }
}


__global__ void fftshift_kernel(hipfftDoubleComplex* data, hipfftDoubleComplex* temp, int width, int height, int shiftX, int shiftY) {
    extern __shared__ hipfftDoubleComplex shared_data[];
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    if (x < width && y < height) {
        shared_data[tid] = data[y * width + x];
    }
    __syncthreads();

    if (x < width && y < height) {
        int new_i = (y + shiftY) % height;
        int new_j = (x + shiftX) % width;
        temp[new_i * width + new_j] = shared_data[tid];
    }
}


void fftshift(thrust::device_vector<hipfftDoubleComplex>& data, int width, int height) {
    thrust::device_vector<hipfftDoubleComplex> temp(data.size());

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    int shiftX = width / 2;
    int shiftY = height / 2;
    size_t sharedMemSize = threadsPerBlock.x * threadsPerBlock.y * sizeof(hipfftDoubleComplex);

    hipStream_t stream;
    hipStreamCreate(&stream);

    fftshift_kernel<<<blocksPerGrid, threadsPerBlock, sharedMemSize, stream>>>(thrust::raw_pointer_cast(data.data()),
                                                        thrust::raw_pointer_cast(temp.data()),
                                                        width, height, shiftX, shiftY);

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    data = temp;
}



void uniformImage(const std::vector<std::complex<double>>& visibilities,
                  const std::vector<double>& u, const std::vector<double>& v,
                  int image_size, std::vector<double>& image) {
    double max_uv = *std::max_element(u.begin(), u.end());
    double pixel_resolution = (0.20 / max_uv) / 3;
    double uv_resolution = 1 / (image_size * pixel_resolution);
    double uv_max = uv_resolution * image_size / 2;
    double grid_res = 2 * uv_max / image_size;

    std::vector<hipfftDoubleComplex> host_visibility_grid(image_size * image_size, make_hipDoubleComplex(0.0, 0.0));

    thrust::device_vector<hipfftDoubleComplex> d_visibility_grid(image_size * image_size, make_hipDoubleComplex(0.0, 0.0));
    thrust::device_vector<double> d_u = u;
    thrust::device_vector<double> d_v = v;

    int threadsPerBlock = 256;
    size_t sharedMemSize = threadsPerBlock * (sizeof(double) * 2 + sizeof(hipfftDoubleComplex));
    size_t chunk_size = image_size * image_size;
    size_t num_chunks = (visibilities.size() + chunk_size - 1) / chunk_size;

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    for (size_t chunk = 0; chunk < num_chunks; ++chunk) {
        size_t start = chunk * chunk_size;
        size_t end = std::min(start + chunk_size, visibilities.size());

        std::vector<hipfftDoubleComplex> vis_chunk_cufft;
        for (size_t i = start; i < end; ++i) {
            vis_chunk_cufft.push_back(make_hipDoubleComplex(visibilities[i].real(), visibilities[i].imag()));
        }

        thrust::device_vector<hipfftDoubleComplex> d_vis_chunk = vis_chunk_cufft;

        int blocksPerGrid = (vis_chunk_cufft.size() + threadsPerBlock - 1) / threadsPerBlock;
        mapVisibilities<<<blocksPerGrid, threadsPerBlock, sharedMemSize, stream1>>>(thrust::raw_pointer_cast(d_visibility_grid.data()),
                                                            thrust::raw_pointer_cast(d_vis_chunk.data()),
                                                            thrust::raw_pointer_cast(d_u.data()) + start,
                                                            thrust::raw_pointer_cast(d_v.data()) + start,
                                                            uv_max, grid_res, image_size, vis_chunk_cufft.size());
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipStreamSynchronize(stream1));
    }

    fftshift(d_visibility_grid, image_size, image_size);

    hipfftHandle plan;
    hipfftPlan2d(&plan, image_size, image_size, HIPFFT_Z2Z);
    hipfftSetStream(plan, stream2);
    hipfftExecZ2Z(plan, thrust::raw_pointer_cast(d_visibility_grid.data()), thrust::raw_pointer_cast(d_visibility_grid.data()), HIPFFT_BACKWARD);
    hipfftDestroy(plan);

    fftshift(d_visibility_grid, image_size, image_size);

    thrust::host_vector<hipfftDoubleComplex> h_output_grid = d_visibility_grid;

    double max_value = 0.0;
    for (size_t i = 0; i < h_output_grid.size(); ++i) {
        if (abs(h_output_grid[i].x) > max_value) {
            max_value = abs(h_output_grid[i].x);
        }
    }

    image.resize(image_size * image_size);
    for (size_t i = 0; i < image.size(); ++i) {
        image[i] = h_output_grid[i].x / max_value;
    }

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
}