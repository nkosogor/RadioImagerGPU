#include "hip/hip_runtime.h"
#include "config.hpp"
#include "compute.hpp"
#include <hipfft/hipfft.h>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include <hip/hip_runtime.h>

/**
 * @brief Check CUDA error and print a message if an error occurs.
 * 
 * @param err CUDA error code.
 * @param msg Error message to display if an error occurs.
 */
void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << " Error: " << hipGetErrorString(err) << "\n";
        exit(EXIT_FAILURE);
    }
}

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

// This atomicAdd is required if your GPU compute capability is less than 6.0
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif



/**
 * @brief Perform a 2D FFT shift operation on the given data.
 * 
 * @param data Input data to be shifted.
 * @param temp Temporary data storage for shifting.
 * @param width Width of the data array.
 * @param height Height of the data array.
 * @param shiftX Amount to shift in the X direction.
 * @param shiftY Amount to shift in the Y direction.
 */
__global__ void fftshift_kernel(hipfftDoubleComplex* data, hipfftDoubleComplex* temp, int width, int height, int shiftX, int shiftY) {
    extern __shared__ hipfftDoubleComplex shared_data[];
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    if (x < width && y < height) {
        shared_data[tid] = data[y * width + x];
    }
    __syncthreads();

    if (x < width && y < height) {
        int new_i = (y + shiftY) % height;
        int new_j = (x + shiftX) % width;
        temp[new_i * width + new_j] = shared_data[tid];
    }
}

/**
 * @brief Perform a 2D FFT shift on a thrust::device_vector.
 * 
 * @param data Data to be shifted.
 * @param width Width of the data array.
 * @param height Height of the data array.
 */
void fftshift(thrust::device_vector<hipfftDoubleComplex>& data, int width, int height) {
    thrust::device_vector<hipfftDoubleComplex> temp(data.size());

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    int shiftX = width / 2;
    int shiftY = height / 2;
    size_t sharedMemSize = threadsPerBlock.x * threadsPerBlock.y * sizeof(hipfftDoubleComplex);

    hipStream_t stream;
    hipStreamCreate(&stream);

    fftshift_kernel<<<blocksPerGrid, threadsPerBlock, sharedMemSize, stream>>>(thrust::raw_pointer_cast(data.data()),
                                                        thrust::raw_pointer_cast(temp.data()),
                                                        width, height, shiftX, shiftY);

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    data = temp;
}


/**
 * @brief Map visibilities to a grid for multiple directions (batches).
 * 
 * @param grid Output grid to store the mapped visibilities.
 * @param visibilities Input visibilities to map.
 * @param u U coordinates of visibilities.
 * @param v V coordinates of visibilities.
 * @param uv_max Maximum UV coordinate value.
 * @param grid_res Resolution of the grid.
 * @param image_size Size of the output image.
 * @param num_visibilities Number of visibilities.
 * @param num_directions Number of directions.
 */
__global__ void mapVisibilitiesMultiDir(hipfftDoubleComplex* grid, const hipfftDoubleComplex* visibilities, const double* u, const double* v, double uv_max, double grid_res, int image_size, int num_visibilities, int num_directions) {
    extern __shared__ double shared_mem[];
    double* shared_u = shared_mem;
    double* shared_v = shared_u + blockDim.x;
    hipfftDoubleComplex* shared_vis = (hipfftDoubleComplex*)(shared_v + blockDim.x);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int dir_idx = blockIdx.y;

    if (dir_idx >= num_directions) return;

    if (idx < num_visibilities) {
        shared_u[threadIdx.x] = u[dir_idx * num_visibilities + idx];
        shared_v[threadIdx.x] = v[dir_idx * num_visibilities + idx];
        shared_vis[threadIdx.x] = visibilities[dir_idx * num_visibilities + idx];
    }
    __syncthreads();

    if (idx < num_visibilities) {
        // Skip adding visibility if both u and v are zero
        if (shared_u[threadIdx.x] == 0.0 && shared_v[threadIdx.x] == 0.0) {
            return;
        }

        int i_index = static_cast<int>((shared_u[threadIdx.x] + uv_max) / grid_res);
        int j_index = static_cast<int>((shared_v[threadIdx.x] + uv_max) / grid_res);
        i_index = (i_index + image_size) % image_size;
        j_index = (j_index + image_size) % image_size;

        if (i_index < image_size && j_index < image_size) {
            atomicAdd(&grid[dir_idx * image_size * image_size + i_index * image_size + j_index].x, shared_vis[threadIdx.x].x);
            atomicAdd(&grid[dir_idx * image_size * image_size + i_index * image_size + j_index].y, shared_vis[threadIdx.x].y);
        }
    }
}

/**
 * @brief Generate a uniform image from visibilities using FFT.
 * 
 * @param visibilities_batch Batch of visibilities for multiple directions.
 * @param u_batch U coordinates for multiple directions.
 * @param v_batch V coordinates for multiple directions.
 * @param image_size Size of the output image.
 * @param images Output images.
 */
void uniformImage(const std::vector<std::vector<std::complex<double>>>& visibilities_batch,
                  const std::vector<std::vector<double>>& u_batch, const std::vector<std::vector<double>>& v_batch,
                  int image_size, std::vector<std::vector<double>>& images, bool use_predefined_params) {
    int num_batches = visibilities_batch.size();
    images.resize(num_batches);

    thrust::device_vector<hipfftDoubleComplex> d_visibility_grid(num_batches * image_size * image_size, make_hipDoubleComplex(0.0, 0.0));
    thrust::device_vector<double> d_u(num_batches * u_batch[0].size());
    thrust::device_vector<double> d_v(num_batches * v_batch[0].size());

    for (int b = 0; b < num_batches; ++b) {
        thrust::copy(u_batch[b].begin(), u_batch[b].end(), d_u.begin() + b * u_batch[0].size());
        thrust::copy(v_batch[b].begin(), v_batch[b].end(), d_v.begin() + b * v_batch[0].size());
    }

    double max_uv = use_predefined_params ? config::PREDEFINED_MAX_UV : *std::max_element(u_batch[0].begin(), u_batch[0].end());
    double pixel_resolution = (0.20 / max_uv) / 3;
    double uv_resolution = 1 / (image_size * pixel_resolution);
    double uv_max = uv_resolution * image_size / 2;
    double grid_res = 2 * uv_max / image_size;

    int threadsPerBlock = 256;
    size_t sharedMemSize = threadsPerBlock * (sizeof(double) * 2 + sizeof(hipfftDoubleComplex));
    size_t chunk_size = image_size * image_size;
    size_t num_chunks = (visibilities_batch[0].size() + chunk_size - 1) / chunk_size;

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    for (size_t chunk = 0; chunk < num_chunks; ++chunk) {
        size_t start = chunk * chunk_size;
        size_t end = std::min(start + chunk_size, visibilities_batch[0].size());

        std::vector<hipfftDoubleComplex> vis_chunk_cufft;
        for (int b = 0; b < num_batches; ++b) {
            for (size_t i = start; i < end; ++i) {
                vis_chunk_cufft.push_back(make_hipDoubleComplex(visibilities_batch[b][i].real(), visibilities_batch[b][i].imag()));
            }
        }

        thrust::device_vector<hipfftDoubleComplex> d_vis_chunk = vis_chunk_cufft;

        dim3 blocksPerGrid((end - start + threadsPerBlock - 1) / threadsPerBlock, num_batches);
        mapVisibilitiesMultiDir<<<blocksPerGrid, threadsPerBlock, sharedMemSize, stream1>>>(thrust::raw_pointer_cast(d_visibility_grid.data()),
                                                                                            thrust::raw_pointer_cast(d_vis_chunk.data()),
                                                                                            thrust::raw_pointer_cast(d_u.data()),
                                                                                            thrust::raw_pointer_cast(d_v.data()),
                                                                                            uv_max, grid_res, image_size, end - start, num_batches);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipStreamSynchronize(stream1));
    }

    for (int b = 0; b < num_batches; ++b) {
        thrust::device_vector<hipfftDoubleComplex> d_visibility_grid_batch(d_visibility_grid.begin() + b * image_size * image_size, d_visibility_grid.begin() + (b + 1) * image_size * image_size);

        fftshift(d_visibility_grid_batch, image_size, image_size);

        hipfftHandle plan;
        hipfftPlan2d(&plan, image_size, image_size, HIPFFT_Z2Z);
        hipfftSetStream(plan, stream2);
        hipfftExecZ2Z(plan, thrust::raw_pointer_cast(d_visibility_grid_batch.data()), thrust::raw_pointer_cast(d_visibility_grid_batch.data()), HIPFFT_BACKWARD);
        hipfftDestroy(plan);

        fftshift(d_visibility_grid_batch, image_size, image_size);

        thrust::host_vector<hipfftDoubleComplex> h_output_grid = d_visibility_grid_batch;

        double max_value = 0.0;
        for (size_t i = 0; i < h_output_grid.size(); ++i) {
            if (abs(h_output_grid[i].x) > max_value) {
                max_value = abs(h_output_grid[i].x);
            }
        }

        images[b].resize(image_size * image_size);
        for (size_t i = 0; i < images[b].size(); ++i) {
            images[b][i] = h_output_grid[i].x / max_value;
        }
    }

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
}

/**
 * @brief CUDA kernel to compute UVW coordinates from XYZ coordinates for multiple directions.
 * 
 * @param x_m X coordinates of the antennas.
 * @param y_m Y coordinates of the antennas.
 * @param z_m Z coordinates of the antennas.
 * @param HAs Hour angles for multiple directions.
 * @param Decs Declinations for multiple directions.
 * @param u Output U coordinates.
 * @param v Output V coordinates.
 * @param w Output W coordinates.
 * @param N Number of antennas.
 * @param num_directions Number of directions.
 */
__global__ void computeUVWKernel(const double* x_m, const double* y_m, const double* z_m, 
                                 const double* HAs, const double* Decs, 
                                 double* u, double* v, double* w, int N, int num_directions) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int dir_idx = blockIdx.y;

    if (dir_idx >= num_directions || idx >= N * (N - 1) / 2) return;

    double HA = HAs[dir_idx];
    double Dec = Decs[dir_idx];

    // Calculate the baseline indices
    int i = static_cast<int>(sqrt(2 * idx + 0.25) - 0.5);
    int j = idx - i * (i + 1) / 2;

    if (i < N && j < N) {
        double dx = x_m[j] - x_m[i];
        double dy = y_m[j] - y_m[i];
        double dz = z_m[j] - z_m[i];

        double u_ij = dx * sin(HA) + dy * cos(HA);
        double v_ij = -dx * sin(Dec) * cos(HA) + dy * sin(Dec) * sin(HA) + dz * cos(Dec);
        double w_ij = dx * cos(Dec) * cos(HA) - dy * cos(Dec) * sin(HA) + dz * sin(Dec);

        int index = dir_idx * N * (N - 1) / 2 + idx;
        u[index] = u_ij;
        v[index] = v_ij;
        w[index] = w_ij;
    }
}

/**
 * @brief Compute UVW coordinates from XYZ coordinates for multiple directions.
 * 
 * @param x_m X coordinates of the antennas.
 * @param y_m Y coordinates of the antennas.
 * @param z_m Z coordinates of the antennas.
 * @param HAs Hour angles for multiple directions.
 * @param Decs Declinations for multiple directions.
 * @param u Output U coordinates for multiple directions.
 * @param v Output V coordinates for multiple directions.
 * @param w Output W coordinates for multiple directions.
* @param use_predefined_params Flag to determine if predefined parameters should be used.
 */
void computeUVW(const std::vector<double>& x_m, const std::vector<double>& y_m, const std::vector<double>& z_m, 
                const std::vector<double>& HAs, const std::vector<double>& Decs, 
                std::vector<std::vector<double>>& u, std::vector<std::vector<double>>& v, std::vector<std::vector<double>>& w) {
    int N = x_m.size();
    int num_directions = HAs.size();
    int num_baselines = N * (N - 1) / 2;

    // Resize output vectors
    u.resize(num_directions, std::vector<double>(num_baselines));
    v.resize(num_directions, std::vector<double>(num_baselines));
    w.resize(num_directions, std::vector<double>(num_baselines));

    thrust::device_vector<double> d_x_m = x_m;
    thrust::device_vector<double> d_y_m = y_m;
    thrust::device_vector<double> d_z_m = z_m;
    thrust::device_vector<double> d_HAs = HAs;
    thrust::device_vector<double> d_Decs = Decs;
    thrust::device_vector<double> d_u(num_directions * num_baselines);
    thrust::device_vector<double> d_v(num_directions * num_baselines);
    thrust::device_vector<double> d_w(num_directions * num_baselines);

    int threadsPerBlock = 256;
    dim3 blocksPerGrid((num_baselines + threadsPerBlock - 1) / threadsPerBlock, num_directions);

    computeUVWKernel<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(d_x_m.data()), 
                                                         thrust::raw_pointer_cast(d_y_m.data()), 
                                                         thrust::raw_pointer_cast(d_z_m.data()), 
                                                         thrust::raw_pointer_cast(d_HAs.data()), 
                                                         thrust::raw_pointer_cast(d_Decs.data()), 
                                                         thrust::raw_pointer_cast(d_u.data()), 
                                                         thrust::raw_pointer_cast(d_v.data()), 
                                                         thrust::raw_pointer_cast(d_w.data()), N, num_directions);

    hipDeviceSynchronize();

    for (int d = 0; d < num_directions; ++d) {
        thrust::copy(d_u.begin() + d * num_baselines, d_u.begin() + (d + 1) * num_baselines, u[d].begin());
        thrust::copy(d_v.begin() + d * num_baselines, d_v.begin() + (d + 1) * num_baselines, v[d].begin());
        thrust::copy(d_w.begin() + d * num_baselines, d_w.begin() + (d + 1) * num_baselines, w[d].begin());
    }
}