#include "hip/hip_runtime.h"
// src/main.cu
#include "config.hpp"
#include "compute.hpp"
#include "data_io.hpp"
#include <iostream>
#include <vector>
#include <complex>
#include <chrono>
#include <fstream>
#include <cmath>  // For M_PI
#include <filesystem>  // For creating directories
#include <argparse/argparse.hpp>

namespace fs = std::filesystem;


/**
 * @brief Main function to compute UVW coordinates, perform imaging, and save results.
 * 
 * @param argc Argument count.
 * @param argv Argument vector.
 * @return int Exit status of the program.
 */
int main(int argc, char* argv[]) {
    config::load_config("config.json");
    
    argparse::ArgumentParser program("RadioImager");

    program.add_argument("--input")
        .default_value(std::string("data/xyz_coordinates.csv"))
        .help("Path to the input CSV file with XYZ coordinates.");

    program.add_argument("--directions")
        .default_value(std::string("data/directions.csv"))
        .help("Path to the directions CSV file with HAs and Decs.");

    program.add_argument("--use_predefined_params")
        .default_value(std::string("true"))
        .help("Use predefined UVW parameters (default: true).");

    program.add_argument("--output_uvw")
        .default_value(std::string("true"))
        .help("Output UVW coordinates (default: true).");

    program.add_argument("--uvw_dir")
        .default_value(std::string("data/uvw_coordinates"))
        .help("Directory to save UVW coordinates.");

    program.add_argument("--image_dir")
        .default_value(std::string("data/images_gpu"))
        .help("Directory to save images.");

    program.add_argument("--save_images")
        .default_value(std::string("true"))
        .help("Save images (default: true).");

    try {
        program.parse_args(argc, argv);
    } catch (const std::runtime_error& err) {
        std::cerr << err.what() << std::endl;
        std::cerr << program;
        return 1;
    }

    const std::string input_path = program.get<std::string>("--input");
    const std::string directions_path = program.get<std::string>("--directions");
    const std::string use_predefined_params_str = program.get<std::string>("--use_predefined_params");
    const bool use_predefined_params = (use_predefined_params_str == "true");
    const std::string output_uvw_str = program.get<std::string>("--output_uvw");
    const bool output_uvw = (output_uvw_str == "true");
    const std::string uvw_dir = program.get<std::string>("--uvw_dir");
    const std::string image_dir = program.get<std::string>("--image_dir");
    const std::string save_images_str = program.get<std::string>("--save_images");
    const bool save_images = (save_images_str == "true");

    std::vector<double> HAs, Decs;
    readDirections(directions_path, HAs, Decs);

    const int image_size = config::IMAGE_SIZE;
    std::vector<double> x_m, y_m, z_m;
    std::vector<std::vector<double>> u, v, w;

    readXYZCoordinates(input_path, x_m, y_m, z_m);

    if (x_m.empty() || y_m.empty() || z_m.empty()) {
        std::cerr << "Error: No data read from file.\n";
        return 1;
    }

    auto start_uvw = std::chrono::high_resolution_clock::now();
    computeUVW(x_m, y_m, z_m, HAs, Decs, u, v, w);
    auto stop_uvw = std::chrono::high_resolution_clock::now();
    auto duration_uvw = std::chrono::duration_cast<std::chrono::milliseconds>(stop_uvw - start_uvw);
    std::cout << "UVW computation complete. Execution time: " << duration_uvw.count() << " ms\n";

    if (output_uvw) {
        saveUVWCoordinates(u, v, w, uvw_dir);
    }

    int num_batches = HAs.size();
    std::vector<std::vector<std::complex<double>>> visibilities(num_batches, std::vector<std::complex<double>>(u[0].size(), std::complex<double>(1, 0)));
    std::vector<std::vector<double>> images;

    auto start = std::chrono::high_resolution_clock::now();
    uniformImage(visibilities, u, v, image_size, images, use_predefined_params);
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    std::cout << "Imaging complete. Execution time: " << duration.count() << " ms\n";

    std::ofstream log_file("output.log", std::ios_base::app);
    log_file << "UVW computation time: " << duration_uvw.count() << " ms\n";
    log_file << "Imaging time: " << duration.count() << " ms\n";
    log_file.close();

    if (save_images) {
        saveImages(images, image_size, image_dir);
    }

    // Reset the GPU
    hipDeviceReset();

    return 0;
}