#include "hip/hip_runtime.h"
#include "imaging.hpp"
#include <hipfft/hipfft.h>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <algorithm>
#include <vector>

// Helper function to replace std::clamp not available in C++11
template<typename T>
T clamp(T val, T minVal, T maxVal) {
    return std::max(minVal, std::min(val, maxVal));
}

// Function to perform circular shift
template<typename T>
void circular_shift(std::vector<T>& data, int image_size) {
    int shift = image_size / 2;
    for (int i = 0; i < shift; ++i) {
        for (int j = 0; j < image_size; ++j) {
            std::swap(data[j * image_size + i], data[j * image_size + (i + shift)]);
        }
    }
    for (int i = 0; i < image_size; ++i) {
        for (int j = 0; j < shift; ++j) {
            std::swap(data[j + i * image_size], data[(j + shift) + i * image_size]);
        }
    }
}


void uniformImage(const std::vector<std::complex<double>>& visibilities,
                  const std::vector<double>& u, const std::vector<double>& v,
                  int image_size, std::vector<double>& image) {
    // Calculate pixel resolution and grid parameters
    double max_uv = *std::max_element(u.begin(), u.end());
    double pixel_resolution = (0.20 / max_uv) / 3;
    double uv_resolution = 1 / (image_size * pixel_resolution);
    double uv_max = uv_resolution * image_size / 2;
    double grid_res = 2 * uv_max / image_size;

    // Create visibility grid on host using CUDA compatible types
    std::vector<hipfftDoubleComplex> h_visibility_grid(image_size * image_size, make_hipDoubleComplex(0, 0));

    // Map visibilities to grid
    for (size_t i = 0; i < visibilities.size(); ++i) {
        int i_index = clamp(static_cast<int>((u[i] + uv_max) / grid_res), 0, image_size - 1);
        int j_index = clamp(static_cast<int>((v[i] + uv_max) / grid_res), 0, image_size - 1);
        int index = i_index * image_size + j_index;
        h_visibility_grid[index].x += visibilities[i].real();
        h_visibility_grid[index].y += visibilities[i].imag();
    }

    // Shift the grid to center the zero-frequency component

    circular_shift(h_visibility_grid, image_size);

    // Transfer data to device
    thrust::device_vector<hipfftDoubleComplex> d_visibility_grid = h_visibility_grid;

    // Prepare and execute FFT
    hipfftHandle plan;
    hipfftResult result = hipfftPlan2d(&plan, image_size, image_size, HIPFFT_Z2Z);
    if (result != HIPFFT_SUCCESS) {
        std::cerr << "CUFFT error: Plan creation failed, error code " << result << std::endl;
        return;
    }

    result = hipfftExecZ2Z(plan, thrust::raw_pointer_cast(d_visibility_grid.data()), thrust::raw_pointer_cast(d_visibility_grid.data()), HIPFFT_BACKWARD);
    if (result != HIPFFT_SUCCESS) {
        std::cerr << "CUFFT error: Executing FFT failed, error code " << result << std::endl;
        hipfftDestroy(plan);
        return;
    }

    hipfftDestroy(plan);

    // Transfer the data back to host
    thrust::host_vector<hipfftDoubleComplex> h_output_grid = d_visibility_grid;

    circular_shift(h_output_grid, image_size);  // Shift back if necessary

    // Normalize the FFT output
    double scale = 1.0 / (image_size * image_size);
    image.resize(image_size * image_size);
    for (size_t i = 0; i < image.size(); ++i) {
        image[i] = h_output_grid[i].x * scale; // Real part of the complex number
    }
}


