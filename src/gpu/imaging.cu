#include "hip/hip_runtime.h"
#include "imaging.hpp"
#include <hipfft/hipfft.h>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <vector>
#include <algorithm>
#include <iostream> 

// This atomicAdd is required if your GPU compute capability is less than 6.0
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

__global__ void mapVisibilities(hipfftDoubleComplex* grid, const hipfftDoubleComplex* visibilities, const double* u, const double* v, double uv_max, double grid_res, int image_size, int num_visibilities) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_visibilities) {
        int i_index = static_cast<int>((u[idx] + uv_max) / grid_res);
        int j_index = static_cast<int>((v[idx] + uv_max) / grid_res);
        i_index = (i_index + image_size) % image_size;
        j_index = (j_index + image_size) % image_size;

        atomicAdd(&grid[i_index * image_size + j_index].x, visibilities[idx].x);
        atomicAdd(&grid[i_index * image_size + j_index].y, visibilities[idx].y);
    }
}

__global__ void fftshift_kernel(hipfftDoubleComplex* data, hipfftDoubleComplex* temp, int width, int height, int shiftX, int shiftY) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int new_i = (y + shiftY) % height;
        int new_j = (x + shiftX) % width;
        temp[new_i * width + new_j] = data[y * width + x];
    }
}

void fftshift(thrust::device_vector<hipfftDoubleComplex>& data, int width, int height) {
    thrust::device_vector<hipfftDoubleComplex> temp(data.size());

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    int shiftX = width / 2;
    int shiftY = height / 2;

    fftshift_kernel<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(data.data()),
                                                        thrust::raw_pointer_cast(temp.data()),
                                                        width, height, shiftX, shiftY);

    hipDeviceSynchronize();
    data = temp;
}



void uniformImage(const std::vector<std::complex<double>>& visibilities,
                  const std::vector<double>& u, const std::vector<double>& v,
                  int image_size, std::vector<double>& image) {
    double max_uv = *std::max_element(u.begin(), u.end());
    double pixel_resolution = (0.20 / max_uv) / 3;
    double uv_resolution = 1 / (image_size * pixel_resolution);
    double uv_max = uv_resolution * image_size / 2;
    double grid_res = 2 * uv_max / image_size;

    // Print the calculated parameters
    std::cout << "max_uv: " << max_uv << "\n";
    std::cout << "pixel_resolution: " << pixel_resolution << "\n";
    std::cout << "uv_resolution: " << uv_resolution << "\n";
    std::cout << "uv_max: " << uv_max << "\n";
    std::cout << "grid_res: " << grid_res << "\n";

    std::vector<hipfftDoubleComplex> host_visibility_grid(image_size * image_size, make_hipDoubleComplex(0.0, 0.0));

    // Initialize the visibilities
    for (int i = 0; i < visibilities.size(); ++i) {
        host_visibility_grid[i] = make_hipDoubleComplex(visibilities[i].real(), visibilities[i].imag());
    }

    thrust::device_vector<hipfftDoubleComplex> d_visibility_grid = host_visibility_grid;
    thrust::device_vector<double> d_u = u;
    thrust::device_vector<double> d_v = v;

    int threadsPerBlock = 256;
    int blocksPerGrid = (visibilities.size() + threadsPerBlock - 1) / threadsPerBlock;
    mapVisibilities<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(d_visibility_grid.data()),
                                                        thrust::raw_pointer_cast(d_visibility_grid.data()),
                                                        thrust::raw_pointer_cast(d_u.data()), 
                                                        thrust::raw_pointer_cast(d_v.data()),
                                                        uv_max, grid_res, image_size, visibilities.size());

    hipDeviceSynchronize();

    // Print central part of visibility grid after mapping
    thrust::host_vector<hipfftDoubleComplex> h_visibility_grid = d_visibility_grid;
    std::cout << "Visibility grid after mapping (central part):\n";
    int center = image_size / 2;
    for (int i = center - 2; i <= center + 2; ++i) {
        for (int j = center - 2; j <= center + 2; ++j) {
            int index = i * image_size + j;
            std::cout << "(" << h_visibility_grid[index].x << ", " << h_visibility_grid[index].y << ") ";
        }
        std::cout << "\n";
    }

    // Apply circular shift before FFT
    fftshift(d_visibility_grid, image_size, image_size);

    // Print central part of visibility grid after first shift
    h_visibility_grid = d_visibility_grid;
    std::cout << "Visibility grid after first FFT shift (central part):\n";
    for (int i = center - 2; i <= center + 2; ++i) {
        for (int j = center - 2; j <= center + 2; ++j) {
            int index = i * image_size + j;
            std::cout << "(" << h_visibility_grid[index].x << ", " << h_visibility_grid[index].y << ") ";
        }
        std::cout << "\n";
    }

    hipfftHandle plan;
    hipfftPlan2d(&plan, image_size, image_size, HIPFFT_Z2Z);
    hipfftExecZ2Z(plan, thrust::raw_pointer_cast(d_visibility_grid.data()), thrust::raw_pointer_cast(d_visibility_grid.data()), HIPFFT_BACKWARD);
    hipfftDestroy(plan);

    // Apply circular shift after FFT
    fftshift(d_visibility_grid, image_size, image_size);

    // Print central part of visibility grid after FFT
    h_visibility_grid = d_visibility_grid;
    std::cout << "Visibility grid after FFT (central part):\n";
    for (int i = center - 2; i <= center + 2; ++i) {
        for (int j = center - 2; j <= center + 2; ++j) {
            int index = i * image_size + j;
            std::cout << "(" << h_visibility_grid[index].x << ", " << h_visibility_grid[index].y << ") ";
        }
        std::cout << "\n";
    }

    thrust::host_vector<hipfftDoubleComplex> h_output_grid = d_visibility_grid;

    // Normalize by the maximum value in the grid
    double max_value = 0.0;
    for (size_t i = 0; i < h_output_grid.size(); ++i) {
        if (abs(h_output_grid[i].x) > max_value) {
            max_value = abs(h_output_grid[i].x);
        }
    }

    image.resize(image_size * image_size);
    for (size_t i = 0; i < image.size(); ++i) {
        image[i] = h_output_grid[i].x / max_value; // Real part of the complex number normalized
    }
    
    /*double scale = 1.0 / (image_size * image_size);  // Ensure the normalization matches the CPU implementation
    image.resize(image_size * image_size);
    for (size_t i = 0; i < image.size(); ++i) {
        image[i] = h_output_grid[i].x * scale; // Real part of the complex number
    }*/
}
